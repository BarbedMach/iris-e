
#include <hip/hip_runtime.h>
extern "C" __global__ void ijk(int* C, int* A, int *B) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] += A[i] + B[i];
}

